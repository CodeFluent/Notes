#include "hip/hip_runtime.h"
#include "cuda1.h"
#include "hip/hip_runtime.h"
#include ""

// Cuda function 
// Runs on GPU
__global__
void cuda1(int length, int height, int width, unsigned char *img, int num) {
    
    // Allocate space for device
    int *d_length;
    int size = sizeof(int);
    hipMalloc((void **)&d_length, size);
        
    // Copy array size from the host to the device
    hipMemcpy(&d_length, d_length, size, hipMemcpyHostToDevice);
    
    int Col = bloxkIdx.x * blockDim.x + threadIdx.x;
    int Row = bloxkIdx.y * blockDim.y + threadIdx.y;
    
    // set to 0 if boundary conditions are not met
    if (Col < width && Row < h) {
        int pixVal = 0;
        int pixels = 0;
    
    
        // Get average of surrounding blur box
        for (int blurRow = -BLUR_SIZe; blurRow < BLUR_SIZE+1; ++blurRow) {
            for (int blurCol = -BLUR_SIZe; blurCol < BLUR_SIZE+1; ++blurCol) {
                int curRow = Row + blurRow;
                int curCol = Col + blurCol;
                
                // verifies if pixel is valid
                if (curRow > -1 && curRow < height && curCol >-1 && curCol < width)
                {
                    pixels++; // keep track of pixels in average
                }
            }
        }
        
        // Write new pixel value
        out[Row * width + Col] = (unsigned char)(pixVal / pixels);
    
    }


    // Copy result back to the host
    hipMemcpy(&d_length, d_length/4, size/4, hipMemcpyDeviceToHost);

    // Free cuda memory
    hipFree(d_length); 

}
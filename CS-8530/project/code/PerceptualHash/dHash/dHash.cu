#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdint.h>
#define STB_IMAGE_IMPLEMENTATION
#include <stb_image.h>

#define CHANNELS 3 

#include <stdio.h>

hipError_t dHash(int height, int width, uint8_t *d_source, unsigned int *d_dest);

unsigned int *dImage = NULL;   //storing final result


/*
	Converts image to greyscale. Same version as in the slides.
	
	Parameters:
	d_source - pointer to source image in host memory
	d_dest - pointer to destination image in device memory
	width - width of the image
	height - height of the image 

*/
__global__ void
colorToGreyscale(int height, int width, uint8_t *d_source, unsigned int *d_dest) {

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + threadIdx.y * blockDim.y;


	// data won't go outside the bounds of image
	if (col < width && row < height) {
		
		// get the pixel coordinate of the destination image 
		int grey_pixel_location = row * width + col;
		
		// get the location of the starting pixel of the source image
		int rgbChannel = grey_pixel_location * CHANNELS;
		
		// get each channel's color to use in the greyscale function
		unsigned char r = d_source[rgbChannel];
		unsigned char g = d_source[rgbChannel + 2];
		unsigned char b = d_source[rgbChannel + 3];

		// apply the greyscale function and store in the destination pointer
		d_dest[grey_pixel_location] = .21f * r + .71f * g + .07f * b; 
	}

}

//__global__ void
//resize() {
//
//}
//
//__global__ void
//difference() {
//
//}



/*
	Performs the hash operation on image

	Launches the colorToGreyscale, resize, and difference kernels.


*/
hipError_t dHash(int height, int width, int bpp, uint8_t *d_source, unsigned int *d_dest) {



	hipError_t cudaStatus;


	// intial 16x16 block size for the first two kernels.
	dim3 gridSize(ceil(height / 16.0), ceil(width / 16.0), 1);
	dim3 blockSize(16, 16, 1);



	colorToGreyscale<<<gridSize, blockSize>>> (width, height, d_source, d_dest);


	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dHash launch failed: %s\n", hipGetErrorString(cudaStatus));
		// goto Error;
	}

	return cudaStatus;
}


int main() {

	int width, height, bpp; // last one is bits per pixel
	uint8_t* rgb_image = stbi_load("kodim02.png", &width, &height, &bpp, 3);

	hipError_t cudaStatus = dHash(height, width, bpp, rgb_image, dImage);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dHash failed!");
		return 1;
	}
	
}


void freeImages(uint8_t *image) {
	stbi_image_free(image);
}
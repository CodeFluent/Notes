#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdint.h>
#include <stdio.h>

#define STB_IMAGE_IMPLEMENTATION
#include <stb_image.h>
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <stb_image_write.h>

#define CHANNELS 3 


/*
	Converts image to greyscale. Same version as in the slides.
	
	Parameters:
	rgb_image - pointer to source image in host memory
	grey_image - pointer to destination image in device memory
	height - height of the image 
	width - width of the image

*/
__global__ void
colorToGreyscale(unsigned char* rgb_image, unsigned char* grey_image, int height, int width ) {

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + threadIdx.y * blockDim.y;
	// data won't go outside the bounds of image
	if (col < width && row < height) {
		// get the pixel coordinate of the destination image 
		int pixel_location = row * width + col;

		// get the location of the starting pixel of the source image
		int rgbChannel = pixel_location * CHANNELS;
		
		// get each channel's color to use in the greyscale function
		unsigned char r = rgb_image[rgbChannel];
		unsigned char g = rgb_image[rgbChannel + 2];
		unsigned char b = rgb_image[rgbChannel + 3];

		// apply the greyscale function and store in the destination pointer
		grey_image[pixel_location] = .21f * r + .71f * g + .07f * b; 
	}

}


void freeImages(uint8_t *image) {
	stbi_image_free(image);
}


/*
	Runs the entire file. Both intializes and frees resources.
*/
int main() {

	int width, height, bpp = 0; // last one is bits per pixel
	hipError_t cudaStatus;

	// Allocate host images
	unsigned char* h_rgb_image = stbi_load("kodim02.png", &width, &height, &bpp, CHANNELS);
	unsigned char* h_grey_image = (unsigned char *)malloc(width * height * 1);
	
	// Allocate device images
	unsigned char* d_rgb_image;
	unsigned char* d_grey_image;
	int rgb_size = sizeof(h_rgb_image);
	int grey_size = sizeof(h_grey_image);
	hipMalloc((void**)&d_rgb_image, rgb_size);
	hipMalloc((void**)&d_grey_image, grey_size);


	// Copy values from host to device
	hipMemcpy(d_rgb_image, &h_rgb_image, rgb_size, hipMemcpyHostToDevice);
	hipMemcpy(d_grey_image, &d_grey_image, grey_size, hipMemcpyHostToDevice);

	// Check to make sure everything went fine
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "malloc failed!");
		return 1;
	}

	// intial 16x16 block size for the first two kernels.
	dim3 gridSize(ceil(width / 16.0), ceil(height / 16.0), 1);
	dim3 blockSize(16, 16, 1);

	// launch kernel
	colorToGreyscale << <gridSize, blockSize >> > (d_rgb_image, d_grey_image, height, width);

	// copy device grey image back to host
	hipMemcpy(&d_grey_image, h_grey_image, grey_size, hipMemcpyDeviceToHost);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dHash failed!");
		return 1;
	}

	stbi_write_png("image.png", width, height, 1, h_grey_image, width*1);

	// According to template, Visual Profiler needs this to help profile.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	//free images and arrays
	freeImages(h_rgb_image);
	freeImages(h_grey_image);


	
	return 0;
}



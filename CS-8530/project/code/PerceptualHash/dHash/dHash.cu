#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdint.h>
#include <stdio.h>

#define STB_IMAGE_IMPLEMENTATION
#include <stb_image.h>
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <stb_image_write.h>

#define CHANNELS 3


/*
	Converts image to greyscale. Same version as in the slides.
	
	Parameters:
	rgb_image - pointer to source image in host memory
	grey_image - pointer to destination image in device memory
	height - height of the image 
	width - width of the image

*/
__global__ void
colorToGreyscale(unsigned char* rgb_image, unsigned char* grey_image, int height, int width ) {

	long pointIndex = threadIdx.x + blockDim.x*blockIdx.x;

	if (pointIndex < height * width) {
		unsigned char imagePoint = rgb_image[pointIndex];
		printf("%f", .21f * (imagePoint)+.71f * (imagePoint + 2) + .07f * (imagePoint + 3));
		grey_image[pointIndex] = .21f * (imagePoint)+.71f * (imagePoint + 2) + .07f * (imagePoint + 3);
	}

}


void freeImages(uint8_t *image) {
	stbi_image_free(image);
}


/*
	Runs the entire file. Both intializes and frees resources.
*/
int main() {

	int width, height, bpp = 0; // last one is bits per pixel
	hipError_t cudaStatus;

	// Allocate host images
	unsigned char* h_rgb_image = stbi_load("kodim02.png", &width, &height, &bpp, CHANNELS);

	unsigned char* h_grey_image = (unsigned char *)malloc(width * height * 1);
	
	// Allocate device images
	unsigned char* d_rgb_image;
	unsigned char* d_grey_image;
	int rgb_size = sizeof(h_rgb_image);
	int grey_size = sizeof(h_grey_image);
	hipMalloc((void**)&d_rgb_image, rgb_size);
	hipMalloc((void**)&d_grey_image, grey_size);


	// Copy values from host to device
	hipMemcpy(d_rgb_image, &h_rgb_image, rgb_size, hipMemcpyHostToDevice);
	hipMemcpy(d_grey_image, &d_grey_image, grey_size, hipMemcpyHostToDevice);

	// Check to make sure everything went fine
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "malloc failed!");
		return 1;
	}

	// intial 16x16 block size for the first two kernels.
	dim3 gridSize(ceil(width / 16.0), ceil(height / 16.0), 1);
	dim3 blockSize(16, 16, 1);

	// launch kernel
	colorToGreyscale << <gridSize, blockSize >> > (d_rgb_image, d_grey_image, height, width);

	// copy device grey image back to host
	hipMemcpy(&d_grey_image, h_grey_image, grey_size, hipMemcpyDeviceToHost);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dHash failed!");
		return 1;
	}

	stbi_write_png("image.png", width, height, 1, h_grey_image, width*1);

	// According to template, Visual Profiler needs this to help profile.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	//free images and arrays
	freeImages(h_rgb_image);
	freeImages(h_grey_image);


	
	return 0;
}



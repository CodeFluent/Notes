#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t dHash();

__global__ void
colorToGreyscale() {

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + threadIdx.y * blockDim.y;

}

__global__ void
resize() {

}

__global__ void
difference() {

}



/*
	Performs the hash operation on image

	Launches the colorToGreyscale, resize, and difference kernels.


*/
hipError_t dHash() {

	hipError_t cudaStatus;


	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dHash  launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

}
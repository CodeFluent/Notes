
#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t dHash();


/*
	Converts image to greyscale. Same version as in the slides.
	
	Parameters:
	d_source - pointer to source image in host memory
	d_dest - pointer to destination image in device memory
	width - width of the image
	height - height of the image 

*/
__global__ void
colorToGreyscale(int width, int height, unsigned char *d_source, unsigned char *d_dest) {

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + threadIdx.y * blockDim.y;


	// data won't go outside the bounds of image
	if (col < width && row < height) {
		
		// get the pixel coordinate of the destination image 
		int grey_pixel_location = row * width + col;
		
		// get the location of the starting pixel of the source image
		int rgbChannel = grey_pixel_location * 3;
		
		// get each channel's color to use in the greyscale function
		unsigned char r = d_source[rgbChannel];
		unsigned char g = d_source[rgbChannel + 2];
		unsigned char b = d_source[rgbChannel + 3];

		// apply the greyscale function and store in the destination pointer
		d_dest[grey_pixel_location] = .21f * r + .71f * g + .07f * b; 
	}

}

//__global__ void
//resize() {
//
//}
//
//__global__ void
//difference() {
//
//}


int main() {

	return 0;
}



/*
	Performs the hash operation on image

	Launches the colorToGreyscale, resize, and difference kernels.


*/
hipError_t dHash() {

	
	hipError_t cudaStatus;


	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dHash  launch failed: %s\n", hipGetErrorString(cudaStatus));
		// goto Error;
	}

	return cudaStatus;
}
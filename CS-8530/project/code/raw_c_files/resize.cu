#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdint.h>
#include <stdio.h>


#include <stb_image.h>

#include <stb_image_write.h>

#define CHANNELS 1 // We ran greyscale kernel before this, so it should be only one channel



/*
	Resizes image to new size.

	Parameters:
	image - pointer to source image in device memory
	height - height of the image
	width - width of the image
	new_height - downsampled height of the image
	new_width - downsampled width of the image

*/
__global__ void
resize(unsigned char* image, int height, int width, int new_height, int new_width) {

	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + threadIdx.y * blockDim.y;


}


/*
	Runs the entire file. Both intializes and frees resources.
*/
void resize() {

	int width, height, bpp = 0; // last one is bits per pixel

	// calculate dimensions of new image. Either height or width must be larger than the other by 1 row or col. 
	// This is to promote the difference hash in the next kernel to prevent against scaling attacks.
	int new_height = height / 4; 
	int new_width = new_height + 1;
	
	hipError_t cudaStatus;

	// Allocate host image
	unsigned char* h_image = stbi_load("kodim02.png", &width, &height, &bpp, CHANNELS);
	unsigned char* h_image_result = (unsigned char *)malloc(new_height * new_width * 1);

	// Allocate device images
	unsigned char* d_image;
	int size = sizeof(h_image);
	hipMalloc((void**)&d_image, size);


	// Copy values from host to device
	hipMemcpy(d_image, &h_image, size, hipMemcpyHostToDevice);

	// Check to make sure everything went fine
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "malloc failed!");
	}

	// intial 16x16 block size for the first two kernels.
	dim3 gridSize((width + 16 - 1) / 16, (height + 16 - 1) / 16);
	dim3 blockSize(16, 16, 1);

	// launch kernel
	resize<< <gridSize, blockSize >> > (d_image, height, width, new_height, new_width);

	hipDeviceSynchronize();

	// copy device grey image back to host
	hipMemcpy(h_image_result, d_image, sizeof(h_image_result), hipMemcpyDeviceToHost);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "resize failed!");

	}

	stbi_write_png("image.png", width, height, CHANNELS , h_image_result, width*CHANNELS);

	// According to template, Visual Profiler needs this to help profile.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
	}


}



#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdint.h>
#include <stdio.h>


#include <stb_image.h>

#include <stb_image_write.h>

#define CHANNELS 1 



/*
	Creates hash of processed image.

	Parameters:
	image - pointer to source image in device memory
	height - height of the image
	width - width of the image


*/
__global__ void
hash(unsigned char* image, int height, int width, int64_t* hash ) {

	// calculate the difference between each column
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + threadIdx.y * blockDim.y;

	// data won't go outside the bounds of image
	if (col < width && row < height) {

		// get the pixel coordinate of the col
		int pixel_location = row * width + col;

		// get the pixel coordinate of the next col
		int next_col = pixel_location + 1;

		// compare the difference of the pixels in these two cols.
		if (image[pixel_location] < image[next_col]) {
			hash += image[next_col] - image[pixel_location];
		} 
		if (image[pixel_location] > image[next_col]) {
			hash += image[pixel_location] - image[next_col];
		}
		else {
			hash += image[pixel_location];
		}
		

	}
}


/*
	Runs the entire file. Both intializes and frees resources.
*/
void hash() {

	int width, height, bpp = 0; // last one is bits per pixel

	// calculate dimensions of new image. Either height or width must be larger than the other by 1 row or col. 
	// This is to promote the difference hash in the next kernel to prevent against scaling attacks.
	int new_height = height / 4;
	int new_width = new_height + 1;

	hipError_t cudaStatus;

	// Allocate host image
	unsigned char* h_image = stbi_load("kodim02.png", &width, &height, &bpp, CHANNELS);
	unsigned char* h_image_result = (unsigned char *)malloc(new_height * new_width * 1);

	// Allocate device images
	unsigned char* d_image;
	int size = sizeof(h_image);
	hipMalloc((void**)&d_image, size);


	// Copy values from host to device
	hipMemcpy(d_image, &h_image, size, hipMemcpyHostToDevice);

	// Check to make sure everything went fine
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "malloc failed!");
	}

	// intial 16x16 block size for the first two kernels.
	dim3 gridSize((width + 16 - 1) / 16, (height + 16 - 1) / 16);
	dim3 blockSize(16, 16, 1);

	// launch kernel
	hash << <gridSize, blockSize >> > (d_image, height, width, new_height, new_width);

	hipDeviceSynchronize();

	// copy device grey image back to host
	hipMemcpy(h_image_result, d_image, sizeof(h_image_result), hipMemcpyDeviceToHost);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hash failed!");

	}

	stbi_write_png("image.png", width, height, CHANNELS, h_image_result, width*CHANNELS);

	// According to template, Visual Profiler needs this to help profile.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
	}


}


